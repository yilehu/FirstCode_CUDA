#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""
#include "Initialization.h"
#include "Initialization_GPU.cu"
#include "PrintToFile.h"
#include "MatrixOperation.cu"

void SelectGPU()
{
	int i;
	hipDeviceProp_t prop;
	hipGetDeviceCount(&i);
	if(i==1)
		printf("    There is %d GPU device on your PC.\n",i);
	else
		printf("    There are %d GPU devices on your PC.\n",i);
	hipGetDeviceProperties(&prop,0);
	printf("    Device %d is: %s.  Compute capability: %d.%d, SMs = %d\n",0,prop.name,prop.major,prop.minor,prop.multiProcessorCount);
	printf("    maxThreadsPerBlock = %d, maxThreadsDim = [%d,%d,%d], maxGridSize = [%d,%d,%d]\n",prop.maxThreadsPerBlock,prop.maxThreadsDim[0],prop.maxThreadsDim[1],prop.maxThreadsDim[2],prop.maxGridSize[0],prop.maxGridSize[1],prop.maxGridSize[2]);
	hipSetDevice(0);
	printf("    Device %d is chosen.\n\n",0);
}

double SumDouble(double *Array,int n)
{
	double Sum=0.0;
	for(int i=0;i<n;i++)
	{
		Sum += Array[i];
	}
	return Sum;
}

int  main()
{
	int GridDim = 80,BlockDim = 256;
	SelectGPU();
	printf("This is my first CUDA code.\n");

	//************ �ļ���д �������� ***********//
	char *Directory1,*Directory2;
	Directory1 = "Array.txt";
	Directory2 = "Matrix.txt";

	//************ ��ʱ�� �������� ***********//
	int START_CLOCK,END_CLOCK;
	double Iter_Running_Time,Total_Running_Time;

	//************ CG �������� ***********//
	int IterationNum;
	int n = 20;
	int Bandwidth = 5;

	double *a,*b,*c,*PartialSum;
	double *dev_a,*dev_b,*dev_c,*dev_PartialSum;

	a = (double*)malloc(n*sizeof(double));
	b = (double*)malloc(n*sizeof(double));
	c = (double*)malloc(n*sizeof(double));
	PartialSum = (double*)malloc(GridDim*sizeof(double));
	hipMalloc((void**)&dev_a,n*sizeof(double));
	hipMalloc((void**)&dev_b,n*sizeof(double));
	hipMalloc((void**)&dev_c,n*sizeof(double));
	hipMalloc((void**)&dev_PartialSum,GridDim*sizeof(double));

	InitializeArray(a,n,1.0);
	InitializeArray(b,n,0.0);
	MatrixMultiply_Banded(a,b,n,2*Bandwidth-1,Bandwidth);
	PrintArray(a,Directory1,"a",n);
	PrintArray(b,Directory1,"b",n);

	InitializeArray_GPU<<<GridDim,BlockDim>>>(dev_a,n,1.0);
	InitializeArray_GPU<<<GridDim,BlockDim>>>(dev_b,n,0.0);
	MatrixMultiply_GPU<<<GridDim,BlockDim>>>(dev_a,dev_b,n,2*Bandwidth-1,Bandwidth);
	hipMemcpy(a,dev_a,n*sizeof(double),hipMemcpyDeviceToHost);
	hipMemcpy(b,dev_b,n*sizeof(double),hipMemcpyDeviceToHost);
	//hipMemcpy(dev_a,a,n*sizeof(double),hipMemcpyHostToDevice);
	//hipMemcpy(dev_b,dev_a,n*sizeof(double),hipMemcpyDeviceToDevice);
	PrintArray(a,Directory1,"a",n);
	PrintArray(b,Directory1,"b",n);

	InitializeArray_GPU<<<GridDim,BlockDim>>>(dev_a,n,1.0);
	InitializeArray_GPU<<<GridDim,BlockDim>>>(dev_b,n,1.0);
	InitializeArray_GPU<<<GridDim,BlockDim>>>(dev_c,n,0.0);
	Dotproduct<<<GridDim,BlockDim>>>(dev_a,dev_b,dev_c,n);
	hipMemcpy(c,dev_c,n*sizeof(double),hipMemcpyDeviceToHost);
	PrintArray(c,Directory1,"c",n);
	printf("Dot Product = %lf\n",SumDouble(c,n));

	InitializeArray_GPU<<<GridDim,1>>>(dev_PartialSum,GridDim,0.0);
	Dotproduct_Shared_Reduction<<<GridDim,BlockDim>>>(dev_a,dev_b,dev_PartialSum,n);
	hipMemcpy(PartialSum,dev_PartialSum,GridDim*sizeof(double),hipMemcpyDeviceToHost);
	printf("Dot Product = %lf\n",SumDouble(PartialSum,GridDim));

	InitializeArray_GPU<<<GridDim,1>>>(dev_PartialSum,GridDim,0.0);
	START_CLOCK = clock();
	for(int i=0;i<1000;i++)
	{
		Pi_Shared_Reduction<<<GridDim,BlockDim>>>(dev_PartialSum,1000000000);
	}
	hipMemcpy(PartialSum,dev_PartialSum,GridDim*sizeof(double),hipMemcpyDeviceToHost);
	END_CLOCK = clock();
	Total_Running_Time = END_CLOCK - START_CLOCK;
	printf("Pi = %22.18lf\n",4.0*SumDouble(PartialSum,GridDim));
	printf("Time = %16.12lf s\n",(double)(END_CLOCK - START_CLOCK)/CLOCKS_PER_SEC);

	return 0;
}
